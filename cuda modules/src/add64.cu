
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif


__global__ void add32(double* A, double* B, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = A[idx] + B[idx]; 
}
	
#ifdef __cplusplus
}
#endif