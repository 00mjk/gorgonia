
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void sigmoid32(float* A, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = 1 / (1 + pow((float)(M_E), (-1 * A[idx])));
}
	
#ifdef __cplusplus
}
#endif