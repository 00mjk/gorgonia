
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif


__global__ void neg64(double* A, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = -A[idx];
}
	
#ifdef __cplusplus
}
#endif